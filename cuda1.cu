
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void add(int a, int b, int *c)//kernel函数，在gpu上运行。
{
    *c = a + b;
}

int main()
{
    int c;
    int *dev_c;
    hipMalloc((void**)&dev_c, sizeof(int));//分配gpu的内存，第一个参数指向新分配内存的地址，第二个参数是分配内存的大小。
    add<<<1,1>>>(2, 7, dev_c);//调用kernel函数，<<<1,1>>>指gpu启动1个线程块，每个线程块中有1个线程。
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);//将gpu上的数据复制到主机上，
	                                                           //即从dev_c指向的存储区域中将sizeof(int)个字节复制到&c指向的存储区域。
    cout << "2 + 7 = " << c << endl;
    hipFree(dev_c);//释放cudaMalloc分配的内存。
    return 0;
}